#include<iostream>
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#define BLOCK_SIZE 64
using namespace std;

void fill_array(int *arr,int size){
    for(int i = 0;i < size; i++){
        arr[i] = rand() % 100;
    }
}

void add_cpu(int *arr1, int *arr2, int *result, int size){
    for(int i = 0;i < size; i++){
        result[i] = arr1[i] + arr2[i];
    }
}

void print_matrix(int *arr, int size){
    for(int i = 0; i < size; i++){
        cout << arr[i] << " ";
    }
    cout << endl;
}

__global__ void add(int *arr1, int *arr2, int *arr3,int size){
    int block_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(block_id < size){
        arr3[block_id] = arr1[block_id] + arr2[block_id];
    }
}

int main(){
    hipEvent_t start,stop;
    float elapsedTime,elapsedTime2;
    
    int *arr1_cpu,*arr2_cpu,*result_cpu;
    int size;
    cout << "Enter size of vector: ";
    cin >> size;

    arr1_cpu = new int[size];
    arr2_cpu = new int[size];
    result_cpu = new int[size];

    fill_array(arr1_cpu,size);
    // cout << "Array 1: ";
    // print_matrix(arr1_cpu,size);
    fill_array(arr2_cpu,size);
    // cout << "Array 2: ";
    // print_matrix(arr2_cpu,size);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    add_cpu(arr1_cpu,arr2_cpu,result_cpu,size);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime2,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout << "CPU result:\n";
    // print_matrix(result_cpu,size);
    cout<<"Elapsed Time = "<<elapsedTime2<<" milliseconds" << endl;

    int *arr1_gpu,*arr2_gpu,*result_gpu;
    
    hipMallocManaged(&arr1_gpu, size * sizeof(int));
    hipMallocManaged(&arr2_gpu, size * sizeof(int));
    hipMallocManaged(&result_gpu, size * sizeof(int));

    hipMemcpy(arr1_gpu,arr1_cpu,size * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu,arr2_cpu,size * sizeof(int),hipMemcpyHostToDevice);
    
    dim3 dimGrid(size / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    add<<<dimGrid,dimBlock>>>(arr1_gpu,arr2_gpu,result_gpu,size);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(result_cpu,result_gpu,size * sizeof(int),hipMemcpyDeviceToHost);
    cout << "GPU result:\n";
    // print_matrix(result_cpu,size);
    cout<<"Elapsed Time = "<<elapsedTime<<" milliseconds" << endl;
    hipFree(arr1_gpu);
    hipFree(arr2_gpu);
    hipFree(result_gpu);

    return 0;
}